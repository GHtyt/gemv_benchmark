#include "hip/hip_runtime.h"
#include "cudaCoreGemm.h"

#include <hip/hip_fp16.h>
float run_cuda_kernel(Params& params, int warmup, int iter)
{
    hipStream_t s;
    hipStreamCreate(&s);
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    for (int i = 0; i < warmup; ++i)
    {
        cudaCoreGemmLauncher<half, half>(params, s);
        //cudaCoreGemmDispatcher(params, s);
    }
    hipEventRecord(begin, s);
    for (int i = 0; i < iter; ++i)
    {
        cudaCoreGemmLauncher<half, half>(params, s);
        //cudaCoreGemmDispatcher(params, s);
    }
    hipEventRecord(end, s);
    hipEventSynchronize(end);
    float time;
    hipEventElapsedTime(&time, begin, end);
    hipEventDestroy(begin);
    hipEventDestroy(end);
    hipStreamDestroy(s);
    return time / iter;
}


int main(int argc, char* argv[]) {
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    int warmup = 20;
    int iter = 100;

    
    printf("fp16   bs : %d    IC = %d   OC = %d    ", m, k, n);

    half*    h_act       = (half   *)malloc(m * k * sizeof(half));
    half*    h_weight    = (half   *)malloc(k * n * sizeof(half));
    half*    h_out       = (half   *)malloc(m * n * sizeof(half));

    
    half*    d_act;
    half*    d_weight;
    half*    d_out;

    hipMalloc(&d_act,        m * k             * sizeof(half));
    hipMalloc(&d_weight,     k * n             * sizeof(half));
    hipMalloc(&d_out,        m * n             * sizeof(half));

    for (int i = 0; i < m * k; ++i) {
        h_act[i] = 1;
    }
    for (int i = 0; i < k * n; ++i) {
        h_weight[i] = 1;
    }
    
    hipMemcpy(d_act,        h_act,        m * k             * sizeof(half),     hipMemcpyHostToDevice);
    hipMemcpy(d_weight,     h_weight,     k * n             * sizeof(half),     hipMemcpyHostToDevice);


    Params params{
        d_act, d_weight, 1.0, d_out, m, n, k};
    float time = run_cuda_kernel(params, warmup, iter);
    double gflops = 2.0 * double(m * n * k) / double(1.0e9) / time;
    printf("time : %f   gflops : %f\n", time, gflops);
    hipMemcpy(h_out,        d_out,        m * n             * sizeof(half),     hipMemcpyDeviceToHost);

    
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < 10; ++j) {
            printf("%.1lf ", __half2float(h_out[i*n+j]));
        }
        printf("\n");
    }
    
    return 0;
}